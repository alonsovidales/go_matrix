/***************************************************
 * Module that applay the function sigmoid to all the elements of the matrix
 * Author: Alonso Vidales <alonso.vidales@tras2.es>
 *
 * To be compiled with nvcc -ptx matrix_sigmoid.cu
 * Debug: nvcc -arch=sm_20 -ptx matrix_sigmoid.cu
 *
 **************************************************/

//#include <stdio.h>

#include <hip/hip_runtime.h>
#include <math.h>

#ifdef __cplusplus
extern "C" {
#endif

// CUDA Kernel
__global__ void matrixSigmoid(float* A, int wA, int hA, int width, int finalSize, int matrixSplits)
{
	for (int bx = 0; bx < matrixSplits; bx++) {
		for (int by = 0; by < matrixSplits; by++) {
			int x = threadIdx.x + (bx * wA);
			int y = threadIdx.y + (by * hA);
			int resultPos = y * width + x;

			if (resultPos < finalSize && x < width) {
				//printf("IN Block %d - %d, wA: %d thread %d - %d Val: %f resultPos: %d finalSize: %d\n", x, y, wA, threadIdx.x, threadIdx.y, A[resultPos], resultPos, finalSize);
				A[resultPos] = 1 / (1 + pow(M_E, (double)(-1 * A[resultPos])));
			}
		}
	}
}

#ifdef __cplusplus
}
#endif
