
#include <hip/hip_runtime.h>
/***************************************************
 * Multiply all the elements of a matrix for the elements of the second one
 * Author: Alonso Vidales <alonso.vidales@tras2.es>
 *
 * To be compiled with nvcc -ptx matrix_mult_all.cu
 * Debug: nvcc -arch=sm_20 -ptx matrix_mult_all.cu
 *
 **************************************************/

//#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

// CUDA Kernel
__global__ void matrixMultAll(float* C, float* A, float* B, int width, int resW, int resH, int resultSize)
{
	int x = threadIdx.x + (blockIdx.x * resW);
        int y = threadIdx.y + (blockIdx.y * resH);
        int resultPos = y * width + x;

	if (resultPos < resultSize && x < width) {
		C[resultPos] = A[resultPos] * B[resultPos];
		//printf("Block %d - %d, thread %d - %d Val: %f\n", x, y, threadIdx.x, threadIdx.y, C[resultPos]);
	}
}

#ifdef __cplusplus
}
#endif
