/***************************************************
 * Module that applay the function sigmoid to all the elements of the matrix
 * Author: Alonso Vidales <alonso.vidales@tras2.es>
 *
 * To be compiled with nvcc -ptx matrix_sum_all.cu
 * Debug: nvcc -arch=sm_20 -ptx matrix_sum_all.cu
 *
 **************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

// CUDA Kernel
__global__ void matrixSumAll(double* A, int wA, int size, double* sum)
{
	__shared__ double res[1024];
	res[threadIdx.x] = 0;

	for (int bx = 0; bx < wA; bx++) {
		int pos = (threadIdx.x * wA) + bx;
		if (pos < size) {
			res[threadIdx.x] += A[pos];
			//printf("Thread %d Pos %d Val: %f\n", threadIdx.x, pos, res[threadIdx.x]);
		}
	}
	__syncthreads();
	if(threadIdx.x == 0) {
		for (int i = 1; i < 1024; i++) {
			res[0] += res[i];
		}
		sum[0] = res[0];
	}
}

#ifdef __cplusplus
}
#endif
