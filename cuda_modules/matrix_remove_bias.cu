
#include <hip/hip_runtime.h>
/***************************************************
 * Module that negs all the elements on a matrix
 * Author: Alonso Vidales <alonso.vidales@tras2.es>
 *
 * To be compiled with nvcc -ptx matrix_remove_bias.cu
 * Debug: nvcc -arch=sm_20 -ptx matrix_remove_bias.cu
 *
 **************************************************/

//#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

// CUDA Kernel
__global__ void matrixRemoveBias(float* C, float* A, int wA, int hA, int width, int finalSize, int matrixSplits)
{
	for (int bx = 0; bx < matrixSplits; bx++) {
		for (int by = 0; by < matrixSplits; by++) {
			int x = threadIdx.x + (bx * wA);
			int y = threadIdx.y + (by * hA);
			int resultPos = y * width + x;

			if (resultPos < finalSize && x <  width) {
				C[resultPos] = A[resultPos + (resultPos / width + 1)];
				//printf("Block %d - %d, thread %d - %d Val: %f Pos: %d Row: %d\n", x, y, threadIdx.x, threadIdx.y, C[resultPos], resultPos, resultPos + (resultPos / width + 1));
			}
			//printf("Block %d - %d, thread %d - %d | %d %d %d %d\n", x, y, threadIdx.x, threadIdx.y, resultPos, finalSize, x, width);
		}
	}
}

#ifdef __cplusplus
}
#endif
